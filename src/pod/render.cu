#include "hip/hip_runtime.h"
#include "render.cuh"

namespace cuda_device
{
    __device__ Color ray_color_device(
        const Ray &r,
        int depth,
        const Hittable *world,
        const BVHNode *bvh_nodes,
        int world_size)
    {
        if (depth <= 0)
        {
            return Color{0, 0, 0};
        }

        HitRecord rec;
        Interval ray_t{0.001f, INFINITY};

        // A more direct way without the top-level hittable dispatcher if the world IS the bvh
        if (hit_bvh(bvh_nodes, world, 0, r, ray_t, rec))
        {
            Ray scattered;
            Color attenuation;
            if (scatter(r, rec, attenuation, scattered))
            {
                return attenuation * ray_color_device(scattered, depth - 1, world, bvh_nodes, world_size);
            }
            return Color{0, 0, 0};
        }

        // If no object was hit, return the background color (sky gradient)
        Vec3 unit_direction = unit_vector(r.direction);
        float t = 0.5f * (unit_direction.y + 1.0f);
        return (1.0f - t) * Color{1.0f, 1.0f, 1.0f} + t * Color{0.5f, 0.7f, 1.0f};
    }
}

extern "C" __global__ void render_kernel(
    unsigned char *framebuffer,
    cuda_device::CameraData cam,
    const cuda_device::Hittable *world,
    const cuda_device::BVHNode *bvh_nodes,
    int world_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid writing outside the framebuffer
    if (i >= PIXEL_WIDTH || j >= PIXEL_HEIGHT)
    {
        return;
    }

    cuda_device::Color pixel_color{0, 0, 0};
    for (int s = 0; s < PIXEL_NEIGHBOR; ++s)
    {
        cuda_device::Ray r = cuda_device::get_ray_device(cam, i, j);
        pixel_color += cuda_device::ray_color_device(r, MAX_DEPTH, world, bvh_nodes, world_size);
    }

    float scale = 1.0f / PIXEL_NEIGHBOR;
    pixel_color *= scale;

    write_color(pixel_color, i, j,
                (char *)framebuffer);
}

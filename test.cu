#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void rand_init(hiprandState *state)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        hiprand_init(0, 0, 0, state); // Explicit seed
    }
}

int main()
{
    hiprandState *d_state;
    hipMalloc((void **)&d_state, sizeof(hiprandState));
    rand_init<<<1, 1>>>(d_state);
    hipDeviceSynchronize();
    hipFree(d_state);
    return 0;
}
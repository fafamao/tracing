#include "legacy/color.h"
#include "legacy/ray.h"
#include "legacy/hittable_list.h"
#include "legacy/sphere.h"
#include "legacy/camera.h"
#include "legacy/material.h"
#include "utility/mem_pool.h"
#include "legacy/bvh_node.h"
#include "legacy/scene.h"
#include "random_number_generator.cuh"
#include "legacy/render.cuh"
#include "pod/generate_scene.h"
#include "pod/bvh_builder.cuh"
#include <hip/hip_runtime.h>
#include <cstring>

// Global random state
__device__ hiprandState *render_rand_state_global;

bool is_gpu_available()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("GPU card is not available.\n");
        return false;
    }
    return deviceCount > 0;
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

void initialize_global_state(int num_pixels)
{
    hiprandState *d_temp_ptr;
    checkCudaErrors(hipMalloc((void **)&d_temp_ptr, num_pixels * sizeof(hiprandState)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(render_rand_state_global), &d_temp_ptr, sizeof(d_temp_ptr)));
}

int main()
{
    // Check GPU availability
    bool is_gpu_ready = is_gpu_available();

    if (is_gpu_ready)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);

        printf("Device %d: %s\n", 0, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Global Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024 * 1024));
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Warp Size: %d\n", prop.warpSize);

        size_t current_stack_size;
        hipDeviceGetLimit(&current_stack_size, hipLimitStackSize);
        std::cout << "Current stack size = " << current_stack_size << " bytes\n";

        // Dimension
        int tx = 16;
        int ty = 16;

        size_t new_stack_size = 16384; // 16 * 1024 bytes
        hipDeviceSetLimit(hipLimitStackSize, new_stack_size);
        std::cout << "Set new stack size = " << new_stack_size << " bytes\n";

        // Construct scene data
        std::vector<cuda_device::Hittable> world_hittable = cuda_device::generate_world();
        cuda_device::BVHBuilder bvh_node(world_hittable);
        std::vector<cuda_device::BVHNode> bvh_nodes = bvh_node.build();
        // Copy scene data to device
        cuda_device::Hittable *d_objects;
        cuda_device::BVHNode *d_nodes;
        checkCudaErrors(hipMalloc(&d_objects, world_hittable.size() * sizeof(cuda_device::Hittable)));
        checkCudaErrors(hipMalloc(&d_nodes, bvh_nodes.size() * sizeof(cuda_device::BVHNode)));
        hipMemcpy(d_objects, world_hittable.data(), world_hittable.size() * sizeof(cuda_device::Hittable), hipMemcpyHostToDevice);
        hipMemcpy(d_nodes, bvh_nodes.data(), bvh_nodes.size() * sizeof(cuda_device::BVHNode), hipMemcpyHostToDevice);

        // Initialize random number of each pixel
        render_init<<<1, 1>>>(PIXEL_WIDTH, PIXEL_HEIGHT);
        checkCudaErrors(hipDeviceSynchronize());
    }
    else
    {
        // Instantiate thread pool
        ThreadPool tp;
        // Memory pool to hold rgb values
        size_t rgb_size = PIXEL_HEIGHT * PIXEL_WIDTH * 3;
        size_t pool_siz = rgb_size * 2;
        MemoryPool mem_pool(pool_siz);
        char *pixel_buffer = mem_pool.allocate(rgb_size);

        // Initialize pixel_buffer all white
        memset(pixel_buffer, 255, rgb_size);

        // Create scene
        hittable_list world;
        generate_scene_host(world);
        // Create camera
        Vec3 camera_origin = Vec3(13, 2, 3);
        Vec3 camera_dest = Vec3(0, 0, 0);
        Vec3 camera_up = Vec3(0, 1, 0);
        Camera camera(camera_origin, camera_dest, camera_up, &tp);
        // Start rendering
        camera.render(world, pixel_buffer);
    }

    printf("Rendering ends\n");

    return 0;
}

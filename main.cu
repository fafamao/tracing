#include "legacy/bvh_node.h"
#include "legacy/camera.h"
#include "legacy/color.h"
#include "legacy/hittable_list.h"
#include "legacy/material.h"
#include "legacy/ray.h"
#include "legacy/render.cuh"
#include "legacy/scene.h"
#include "legacy/sphere.h"
#include "pod/bvh_builder.cuh"
#include "pod/camera_pod.cuh"
#include "pod/generate_scene.h"
#include "pod/render.cuh"
#include "random_number_generator.cuh"
#include "utility/mem_pool.h"
#include <cstring>
#include <hip/hip_runtime.h>

// Global random state
__device__ hiprandState *render_rand_state_global;

bool is_gpu_available()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("GPU card is not available.\n");
        return false;
    }
    return deviceCount > 0;
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func,
                const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
                  << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

void initialize_global_state(int num_pixels)
{
    hiprandState *d_temp_ptr;
    checkCudaErrors(
        hipMalloc((void **)&d_temp_ptr, num_pixels * sizeof(hiprandState)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(render_rand_state_global), &d_temp_ptr,
                                       sizeof(d_temp_ptr)));
}

int main()
{
    // Check GPU availability
    bool is_gpu_ready = is_gpu_available();

    if (is_gpu_ready)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);

        printf("Device %d: %s\n", 0, prop.name);
        printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
        printf("  Total Global Memory: %.2f GB\n",
               prop.totalGlobalMem / (1024.0 * 1024 * 1024));
        printf("  Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Blocks per SM: %d\n", prop.maxBlocksPerMultiProcessor);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Warp Size: %d\n", prop.warpSize);

        size_t current_stack_size;
        hipDeviceGetLimit(&current_stack_size, hipLimitStackSize);
        std::cout << "Current stack size = " << current_stack_size << " bytes\n";

        size_t new_stack_size = 16384; // 16 * 1024 bytes
        hipDeviceSetLimit(hipLimitStackSize, new_stack_size);
        std::cout << "Set new stack size = " << new_stack_size << " bytes\n";

        // Construct scene data
        std::vector<cuda_device::Hittable> world_hittable =
            cuda_device::generate_world();
        cuda_device::BVHBuilder bvh_node(world_hittable);
        std::vector<cuda_device::BVHNode> bvh_nodes = bvh_node.build();
        // For debug purpose
        // bvh_node.print_tree();
        // Copy scene data to device
        cuda_device::Hittable *d_objects;
        cuda_device::BVHNode *d_nodes;
        checkCudaErrors(hipMalloc(&d_objects, world_hittable.size() *
                                                   sizeof(cuda_device::Hittable)));
        checkCudaErrors(
            hipMalloc(&d_nodes, bvh_nodes.size() * sizeof(cuda_device::BVHNode)));
        checkCudaErrors(
            hipMemcpy(d_objects, world_hittable.data(),
                       world_hittable.size() * sizeof(cuda_device::Hittable),
                       hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_nodes, bvh_nodes.data(),
                                   bvh_nodes.size() * sizeof(cuda_device::BVHNode),
                                   hipMemcpyHostToDevice));

        // Construct camera
        cuda_device::Vec3 camera_origin = cuda_device::Vec3{13, 2, 3};
        cuda_device::Vec3 camera_dest = cuda_device::Vec3{0, 0, 0};
        cuda_device::Vec3 camera_up = cuda_device::Vec3{0, 1, 0};
        cuda_device::CameraData camera =
            cuda_device::construct_camera(camera_origin, camera_dest, camera_up);

        // Initialize random state
        initialize_global_state(PIXEL_WIDTH * PIXEL_HEIGHT);

        // Allocate pinned host memory for pixel data transfer
        unsigned char *d_pixel_data;
        unsigned char *h_pixel_data;
        checkCudaErrors(hipMalloc((void **)&d_pixel_data, FRAME_BUFFERING));
        checkCudaErrors(hipHostMalloc((void **)&h_pixel_data, FRAME_BUFFERING));

        // Initialize random number of each pixel
        render_init<<<1, 1>>>(PIXEL_WIDTH, PIXEL_HEIGHT);
        checkCudaErrors(hipDeviceSynchronize());

        // Dimension
        int number_of_thread_x = 16;
        int number_of_thread_y = 16;
        dim3 threadsPerBlock(number_of_thread_x, number_of_thread_y);
        dim3 blocksPerGrid(
            (PIXEL_WIDTH + number_of_thread_x - 1) / number_of_thread_x,
            (PIXEL_HEIGHT + number_of_thread_y - 1) / number_of_thread_y);

        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        checkCudaErrors(hipEventRecord(start));

        render_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_pixel_data, camera, d_objects, d_nodes, world_hittable.size());
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipEventRecord(stop));

        checkCudaErrors(hipEventSynchronize(stop));

        float milliseconds = 0;
        checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

        printf("Kernel execution time: %f ms\n", milliseconds);

        checkCudaErrors(hipEventDestroy(start));
        checkCudaErrors(hipEventDestroy(stop));

        checkCudaErrors(hipMemcpy(h_pixel_data, d_pixel_data, FRAME_BUFFERING, hipMemcpyDeviceToHost));

        generate_ppm_6((char *)h_pixel_data);

        // Free resource
        checkCudaErrors(hipFree(d_objects));
        checkCudaErrors(hipFree(d_nodes));
        checkCudaErrors(hipFree(d_pixel_data));
        checkCudaErrors(hipHostFree(h_pixel_data));
    }
    else
    {
        // Instantiate thread pool
        ThreadPool tp;
        // Memory pool to hold rgb values
        size_t rgb_size = PIXEL_HEIGHT * PIXEL_WIDTH * 3;
        size_t pool_siz = rgb_size * 2;
        MemoryPool mem_pool(pool_siz);
        char *pixel_buffer = mem_pool.allocate(rgb_size);

        // Initialize pixel_buffer all white
        memset(pixel_buffer, 255, rgb_size);

        // Create scene
        hittable_list world;
        generate_scene_host(world);
        // Create camera
        Vec3 camera_origin = Vec3(13, 2, 3);
        Vec3 camera_dest = Vec3(0, 0, 0);
        Vec3 camera_up = Vec3(0, 1, 0);
        Camera camera(camera_origin, camera_dest, camera_up, &tp);
        // Start rendering
        camera.render(world, pixel_buffer);
    }

    printf("Rendering ends\n");

    return 0;
}
